#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "reduce.h"
#include "graph.h"

extern int verbose;
extern int debug;

/****************************************************************

##     ## ######## ##       ########  ######## ########   ######  
##     ## ##       ##       ##     ## ##       ##     ## ##    ## 
##     ## ##       ##       ##     ## ##       ##     ## ##       
######### ######   ##       ########  ######   ########   ######  
##     ## ##       ##       ##        ##       ##   ##         ## 
##     ## ##       ##       ##        ##       ##    ##  ##    ## 
##     ## ######## ######## ##        ######## ##     ##  ######  

*****************************************************************/

/*
Templated swap for queues
*/
template <typename T> void swap(T* &p, T* &q) {
  T* tmp = p;
  p = q;
  q = tmp;
}

/*
Adds edges from F and T to the output srcs and dsts
*/
void add_new_edges(int* f_parents, int* t_parents, int* new_srcs, int* new_dsts, int &new_num_edges, int num_verts) 
{
  double elt = omp_get_wtime();
  if (verbose) printf("Adding Edges to Output ......... ");

  new_num_edges = 0;
  for (int i = 0; i < num_verts; ++i) {
    int u = f_parents[i];
    int w = t_parents[i];

    if (i != u && u != -1) {
      int src = i < u ? i : u;
      int dst = i < u ? u : i;

      new_srcs[new_num_edges] = src;
      new_dsts[new_num_edges] = dst;
      ++new_num_edges;
    }

    // Only add w if it is not the same edge as u
    if (w != i && w != u && w != -1) {
      int src = i < w ? i : w;
      int dst = i < w ? w : i;

      new_srcs[new_num_edges] = src;
      new_dsts[new_num_edges] = dst;
      ++new_num_edges;
    }
  }

  if (verbose) printf("Done: %lf (s)\n", omp_get_wtime() - elt);
  if (debug) printf("\tnum edges after adding F & T: %d\n", new_num_edges);

}

/*
Kernel that adds vertex to the next queue if the in_queue_next flag was set for that vertex
*/
__global__
void update_queue_next(graph* g, int* queue_next, int* queue_next_size, bool* in_queue, bool* in_queue_next) 
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < g->n) {
    in_queue[index] = false;
    if (in_queue_next[index]) {
      int queue_index = atomicAdd(queue_next_size, 1);
      queue_next[queue_index] = index;
    }
  }
}


/********************************************************************************************************************************************

 ######  ########     ###    ##    ## ##    ## #### ##    ##  ######      ######## ########  ######## ########    ########  ########  ######  
##    ## ##     ##   ## ##   ###   ## ###   ##  ##  ###   ## ##    ##        ##    ##     ## ##       ##          ##     ## ##       ##    ## 
##       ##     ##  ##   ##  ####  ## ####  ##  ##  ####  ## ##              ##    ##     ## ##       ##          ##     ## ##       ##       
 ######  ########  ##     ## ## ## ## ## ## ##  ##  ## ## ## ##   ####       ##    ########  ######   ######      ########  ######    ######  
      ## ##        ######### ##  #### ##  ####  ##  ##  #### ##    ##        ##    ##   ##   ##       ##          ##     ## ##             ## 
##    ## ##        ##     ## ##   ### ##   ###  ##  ##   ### ##    ##        ##    ##    ##  ##       ##          ##     ## ##       ##    ## 
 ######  ##        ##     ## ##    ## ##    ## #### ##    ##  ######         ##    ##     ## ######## ########    ########  ##        ###### 

********************************************************************************************************************************************/

/*
Spannning Tree BFS Initialization Kernel
*/
__global__
void st_bfs_init(graph* g, int* root, int* parents, int* levels, int* queue, int* queue_size)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  if (index < g->n) {
    // Starting source is default to max_degree -- can change here if desired
    if (index == *root) {
      parents[index] = index;
      levels[index] = 0;

      int queue_index = atomicAdd(queue_size, 1);
      queue[queue_index] = index;
    }
    else {
      parents[index] = -1;
      levels[index] = -1;
    }
  }
  
  return;
}

/*
Spanning Tree Top Down BFS Iteration Kernel - Updates all neighbors of only queue vertices
*/
 __global__
 void st_td_bfs_level(graph* g, int* parents, int* levels, int level, int* queue, int* queue_size, int* queue_next, int* queue_next_size, bool* in_queue, bool* in_queue_next) 
 {
   int index = (blockIdx.x * blockDim.x) + threadIdx.x;
 
   if (index < *queue_size) {
     int vert = queue[index];
     in_queue[vert] = false;
 
     int degree = out_degree(g, vert);
     int* outs = out_adjs(g, vert);
 
     for (int j = 0; j < degree; ++j) {
       int out = outs[j];
 
       if (parents[out] == -1) {
         parents[out] = vert;
         levels[out] = level;
         
         in_queue_next[out] = true;
       }
     }
   }
 }

/*
Spanning Tree Bottom Up BFS Iteration Kernel - Updates all vertices with first neighbor in prev level
*/
 __global__
 void st_bu_bfs_level(graph* g, int* parents, int* levels, int level, int* queue, int* queue_size, int* queue_next, int* queue_next_size, bool* in_queue, bool* in_queue_next) 
 {
   int index = (blockIdx.x * blockDim.x) + threadIdx.x;
 
   if (index < g->n) {
     int vert = index;
     int prev_level = level - 1;
 
     if (levels[vert] < 0) {
       int degree = out_degree(g, vert);
       int* outs = out_adjs(g, vert);
 
       for (int j = 0; j < degree; ++j) {
         int out = outs[j];
 
         if (levels[out] == prev_level) {
           levels[vert] = level;
           parents[vert] = out;
           in_queue_next[vert] = true;
           break;
         }
       }
     }
   }
 }

/*
Hybrid BFS - Generates spanning tree for G, switching between top-down and bottom-up based on next queue parameters
*/
void spanning_tree(graph* g, int* root, int* parents, int* levels, int* queue, int* queue_size, int* queue_next, int* queue_next_size, bool* in_queue, bool* in_queue_next) 
{
  double elt = omp_get_wtime();
  if (verbose) printf("Running Spanning Tree BFS ...... ");

  int thread_blocks_n = g->n / BLOCK_SIZE + 1;
  int init_queue_size = 0;

  bool using_top_down = true;
  bool already_switched = false;
  if (debug) printf("\n\tStarting with top-down BFS\n");

  st_bfs_init<<<thread_blocks_n, BLOCK_SIZE>>>(g, root, parents, levels, queue, queue_size);

  hipDeviceSynchronize();

  int level = 1;
  while (*queue_size) {    
    // Switch BFS type if necessary
    if (!already_switched) {
      int frontier_size = *queue_size;
      if (using_top_down) {
        double edges_frontier = (double)frontier_size * g->avg_out_degree;
        double edges_remainder = (double)(g->n - frontier_size) * g->avg_out_degree; 
        if ((edges_remainder / ALPHA) < edges_frontier && edges_remainder > 0) {
          if (debug) printf("\tSwitching to bottom-up BFS on level %d\n", level);
          using_top_down = false;
        }
      } else if (((double)g->n / BETA) > frontier_size){
        if (debug) printf("\tSwitching back to top-down BFS on level %d\n", level);
        using_top_down = false;
        already_switched = true;
      }
    }

    // Run iteration of BFS
    if (using_top_down) {
      st_td_bfs_level<<<thread_blocks_n, BLOCK_SIZE>>>(g, parents, levels, level, queue, queue_size, queue_next, queue_next_size, in_queue, in_queue_next);
      hipDeviceSynchronize();
    } else {
      st_bu_bfs_level<<<thread_blocks_n, BLOCK_SIZE>>>(g, parents, levels, level, queue, queue_size, queue_next, queue_next_size, in_queue, in_queue_next);
      hipDeviceSynchronize();
    }

    // Update next queue
    update_queue_next<<<thread_blocks_n, BLOCK_SIZE>>>(g, queue_next, queue_next_size, in_queue, in_queue_next);
    hipDeviceSynchronize();

    // Swap queues
    swap(queue, queue_next);
    swap(in_queue, in_queue_next);
    hipMemcpy(queue_size, queue_next_size, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(queue_next_size, &init_queue_size, sizeof(int), hipMemcpyHostToDevice);

    level++;
  }

  if (verbose) printf("Done: %lf (s)\n", omp_get_wtime() - elt);
}

/**********************************************************************************************************************************************************************************

 ######   #######  ##    ## ##    ## ########  ######  ######## ######## ########      ######   #######  ##     ## ########   #######  ##    ## ######## ##    ## ########  ######  
##    ## ##     ## ###   ## ###   ## ##       ##    ##    ##    ##       ##     ##    ##    ## ##     ## ###   ### ##     ## ##     ## ###   ## ##       ###   ##    ##    ##    ## 
##       ##     ## ####  ## ####  ## ##       ##          ##    ##       ##     ##    ##       ##     ## #### #### ##     ## ##     ## ####  ## ##       ####  ##    ##    ##       
##       ##     ## ## ## ## ## ## ## ######   ##          ##    ######   ##     ##    ##       ##     ## ## ### ## ########  ##     ## ## ## ## ######   ## ## ##    ##     ######  
##       ##     ## ##  #### ##  #### ##       ##          ##    ##       ##     ##    ##       ##     ## ##     ## ##        ##     ## ##  #### ##       ##  ####    ##          ## 
##    ## ##     ## ##   ### ##   ### ##       ##    ##    ##    ##       ##     ##    ##    ## ##     ## ##     ## ##        ##     ## ##   ### ##       ##   ###    ##    ##    ## 
 ######   #######  ##    ## ##    ## ########  ######     ##    ######## ########      ######   #######  ##     ## ##         #######  ##    ## ######## ##    ##    ##     ######  

**********************************************************************************************************************************************************************************/

/*
Connected Components Initialization
*/
__global__
void cc_init(graph* g, int* parents, int* labels, 
             int* queue, int* queue_size, bool* in_queue, bool* in_queue_next)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  // Add all v in G - T to queue
  if (index < g->n){
    labels[index] = index;
    
    queue[index] = index;

    in_queue[index] = true;
    in_queue_next[index] = false;
  }
  
  return;
}

/*
Connected Components Kernel - Updates labels and adds any relevant neighbors to the next queue
*/
__global__
void cc_level(graph* g, int* parents, int* labels, int* queue, int* queue_size, 
              int* queue_next, int* queue_next_size, bool* in_queue, bool* in_queue_next) 
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < *queue_size) {
    int v = queue[index];
    in_queue[v] = false;

    int degree = out_degree(g, v);
    int* outs = out_adjs(g, v);

    bool changed = false;
    for (int j = 0; j < degree; ++j) {
      int u = outs[j];

      if (parents[u] == v || parents[v] == u) continue;

      if (labels[v] > labels[u]) {        
        labels[v] = labels[u];
        changed = true;
      }
    }
    
    if (changed) {
      in_queue_next[v] = true;

      for (int j = 0; j < degree; ++j) {
        int u = outs[j];
        if (parents[u] == v || parents[v] == u) continue;
        in_queue_next[u] = true;
      }
    }
  }
}

/*
Connected Components - Generates labels for every vertex
*/
void connected_components(graph* g, int * parents, int* labels, int* queue, int* queue_size, 
                          int* queue_next, int* queue_next_size, bool* in_queue, bool* in_queue_next) 
{
  double elt = omp_get_wtime();
  if (verbose) printf("Running Connected Components ... ");

  int thread_blocks_n = g->n / BLOCK_SIZE + 1;

  cc_init<<<thread_blocks_n, BLOCK_SIZE>>>(g, parents, labels, queue, queue_size, in_queue, in_queue_next);
  hipMemcpy(queue_size, &g->n, sizeof(int), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  int init_queue_size = 0;
  while (*queue_size) {
    // Run level & update queue next
    cc_level<<<thread_blocks_n, BLOCK_SIZE>>>(g, parents, labels, queue, queue_size, queue_next, queue_next_size, in_queue, in_queue_next);
    hipDeviceSynchronize();

    update_queue_next<<<thread_blocks_n, BLOCK_SIZE>>>(g, queue_next, queue_next_size, in_queue, in_queue_next);
    hipDeviceSynchronize();

    // Swap queues
    swap(queue, queue_next);
    swap(in_queue, in_queue_next);

    hipMemcpy(queue_size, queue_next_size, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(queue_next_size, &init_queue_size, sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
  }

  if (verbose) printf("Done: %lf (s)\n", omp_get_wtime() - elt);
}

/***************************************************************************************************************************************************************

 ######  ########     ###    ##    ## ##    ## #### ##    ##  ######      ########  #######  ########  ########  ######  ########    ########  ########  ######  
##    ## ##     ##   ## ##   ###   ## ###   ##  ##  ###   ## ##    ##     ##       ##     ## ##     ## ##       ##    ##    ##       ##     ## ##       ##    ## 
##       ##     ##  ##   ##  ####  ## ####  ##  ##  ####  ## ##           ##       ##     ## ##     ## ##       ##          ##       ##     ## ##       ##       
 ######  ########  ##     ## ## ## ## ## ## ##  ##  ## ## ## ##   ####    ######   ##     ## ########  ######    ######     ##       ########  ######    ######  
      ## ##        ######### ##  #### ##  ####  ##  ##  #### ##    ##     ##       ##     ## ##   ##   ##             ##    ##       ##     ## ##             ## 
##    ## ##        ##     ## ##   ### ##   ###  ##  ##   ### ##    ##     ##       ##     ## ##    ##  ##       ##    ##    ##       ##     ## ##       ##    ## 
 ######  ##        ##     ## ##    ## ##    ## #### ##    ##  ######      ##        #######  ##     ## ########  ######     ##       ########  ##        ######  

****************************************************************************************************************************************************************/

/*
Spanning Forest BFS Initialization Kernel
*/
__global__
void sf_bfs_init(graph* g, int* labels, int* parents, int* levels, int* queue, int* queue_size, bool* in_queue, bool* in_queue_next)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  if (index < g->n) {
    // If label equals itself then it must be a source since it is the lowest label for component
    if (index == labels[index]) {
      parents[index] = index;
      levels[index] = 0;
      
      in_queue[index] = true;
      in_queue_next[index] = false;

      int queue_index = atomicAdd(queue_size, 1);
      queue[queue_index] = index;
    } else {
      parents[index] = -1;
      levels[index] = -1;

      in_queue[index] = false;
      in_queue_next[index] = false;
    }
  }
  
  return;
}

/*
Spanning Forest Top Down BFS Iteration Kernel - Updates all neighbors of only queue vertices
*/
__global__
void sf_td_bfs_level(graph* g, int* t_parents, int* f_parents, int* levels, int level, int* queue, int* queue_size, int* queue_next, int* queue_next_size, bool* in_queue, bool* in_queue_next) 
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (index < *queue_size) {
    int vert = queue[index];
    in_queue[vert] = false;

    int degree = out_degree(g, vert);
    int* outs = out_adjs(g, vert);

    for (int j = 0; j < degree; ++j) {
      int out = outs[j];

      // Dont use if in T or already has parent
      if (t_parents[vert] == out || t_parents[out] == vert ||
          f_parents[vert] == out || f_parents[out] == vert)
      continue;

      if (f_parents[out] == -1) {
        f_parents[out] = vert;
        levels[out] = level;
        
        in_queue_next[out] = true;
      }
    }
  }
}

/*
Spanning Forest Bottom Up BFS Iteration Kernel - Updates all vertices with first neighbor in prev level
*/
 __global__
 void sf_bu_bfs_level(graph* g, int* t_parents, int* f_parents, int* levels, int level, int* queue, int* queue_size, int* queue_next, int* queue_next_size, bool* in_queue, bool* in_queue_next) 
 {
   int index = (blockIdx.x * blockDim.x) + threadIdx.x;
 
   if (index < g->n) {
     int vert = index;
     int prev_level = level - 1;
 
     if (levels[vert] < 0) {
       int degree = out_degree(g, vert);
       int* outs = out_adjs(g, vert);
 
       for (int j = 0; j < degree; ++j) {
         int out = outs[j];

         // Dont use if in T or already has parent
        if (t_parents[vert] == out || t_parents[out] == vert ||
            f_parents[vert] == out || f_parents[out] == vert)
          continue;
 
         if (levels[out] == prev_level) {
           levels[vert] = level;
           f_parents[vert] = out;
           in_queue_next[vert] = true;
           break;
         }
       }
     }
   }
 }

 /*
 Hybrid BFS - Generates spanning forest for G-T, with labels as src
 Switches between top-down and bottom-up based on next queue parameters
 */
void spanning_forest(graph* g, int* labels, int* t_parents, int* f_parents, int* levels, int* queue, int* queue_size, int* queue_next, int* queue_next_size, bool* in_queue, bool* in_queue_next) 
{
  double elt = omp_get_wtime();
  if (verbose) printf("Running Spanning Forest BFS .... ");

  int thread_blocks_n = g->n / BLOCK_SIZE + 1;
  int init_queue_size = 0;

  bool using_top_down = true;
  bool already_switched = false;

  sf_bfs_init<<<thread_blocks_n, BLOCK_SIZE>>>(g, labels, f_parents, levels, queue, queue_size, in_queue, in_queue_next);

  hipDeviceSynchronize();

  if (debug) printf("\n\tNumber of CC in G - T (source count): %d\n", *queue_size);
  if (debug) printf("\tStarting with top-down BFS\n");

  int level = 1;
  while (*queue_size) {   
    // Switch BFS type if necessary
    if (!already_switched) {
      int frontier_size = *queue_size;
      if (using_top_down) {
        double edges_frontier = (double)frontier_size * g->avg_out_degree;
        double edges_remainder = (double)(g->n - frontier_size) * g->avg_out_degree; 
        if ((edges_remainder / ALPHA) < edges_frontier && edges_remainder > 0) {
          if (debug) printf("\tSwitching to bottom-up BFS on level %d\n", level);
          using_top_down = false;
        }
      } else if (((double)g->n / BETA) > frontier_size){
        if (debug) printf("\tSwitching back to top-down BFS on level %d\n", level);
        using_top_down = false;
        already_switched = true;
      }
    }

    // Run iteration of BFS
    if (using_top_down) {
      sf_td_bfs_level<<<thread_blocks_n, BLOCK_SIZE>>>(g, t_parents, f_parents, levels, level, queue, queue_size, queue_next, queue_next_size, in_queue, in_queue_next);
      hipDeviceSynchronize();
    } else {
      sf_bu_bfs_level<<<thread_blocks_n, BLOCK_SIZE>>>(g, t_parents, f_parents, levels, level, queue, queue_size, queue_next, queue_next_size, in_queue, in_queue_next);
      hipDeviceSynchronize();
    }

    // Update next queue
    update_queue_next<<<thread_blocks_n, BLOCK_SIZE>>>(g, queue_next, queue_next_size, in_queue, in_queue_next);
    hipDeviceSynchronize();

    // Swap queues
    swap(queue, queue_next);
    swap(in_queue, in_queue_next);
    hipMemcpy(queue_size, queue_next_size, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(queue_next_size, &init_queue_size, sizeof(int), hipMemcpyHostToDevice);

    level++;
  }

  if (verbose) printf("Done: %lf (s)\n", omp_get_wtime() - elt);
}

/********************************

##     ##    ###    #### ##    ## 
###   ###   ## ##    ##  ###   ## 
#### ####  ##   ##   ##  ####  ## 
## ### ## ##     ##  ##  ## ## ## 
##     ## #########  ##  ##  #### 
##     ## ##     ##  ##  ##   ### 
##     ## ##     ## #### ##    ##

********************************/

int reduce_graph_gpu(graph* g_host, int* new_srcs, int* new_dsts, int& new_num_edges) {
  double elt = omp_get_wtime();
  if (verbose) printf("Initializing data on GPU ....... ");
  
  // copy graph data to GPU
  graph* g = NULL;
  assert(hipMallocManaged(&g, sizeof(graph)) == hipSuccess);
  hipMemcpy(&g->n, &g_host->n, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(&g->m, &g_host->m, sizeof(long), hipMemcpyHostToDevice);
  hipMemcpy(&g->max_degree, &g_host->max_degree, sizeof(long), hipMemcpyHostToDevice);
  hipMemcpy(&g->max_degree_vert, &g_host->max_degree_vert, sizeof(long), hipMemcpyHostToDevice);
  hipMemcpy(&g->avg_out_degree, &g_host->avg_out_degree, sizeof(double), hipMemcpyHostToDevice);

  int num_verts = g_host->n;
  int num_edges = g_host->m;
  assert(hipMallocManaged(&g->out_adjlist, num_edges*sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&g->out_offsets, (num_verts + 1)*sizeof(long)) == hipSuccess);
  hipMemcpy(g->out_adjlist, g_host->out_adjlist, num_edges*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g->out_offsets, g_host->out_offsets, (num_verts + 1)*sizeof(long), hipMemcpyHostToDevice);
  
  // init root, parents, levels array for spanning tree & forest
  int* root = NULL;
  int* t_parents = NULL;
  int* t_levels = NULL;
  int* f_parents = NULL;
  int* f_levels = NULL;
  assert(hipMallocManaged(&root, sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&t_parents, num_verts*sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&t_levels, num_verts*sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&f_parents, num_verts*sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&f_levels, num_verts*sizeof(int)) == hipSuccess);
  hipMemcpy(root, &g_host->max_degree_vert, sizeof(int), hipMemcpyHostToDevice);

  // init connected components array
  int* labels = NULL;
  assert(hipMallocManaged(&labels, num_verts*sizeof(int)) == hipSuccess);

  // init queue variables
  int init_queue_size = 0;
  int* queue = NULL;
  int* queue_size = NULL;
  int* queue_next = NULL;
  int* queue_next_size = NULL;
  bool* in_queue = NULL;
  bool* in_queue_next = NULL;

  assert(hipMallocManaged(&queue, num_verts*sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&queue_size, sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&queue_next, num_verts*sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&queue_next_size, sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&in_queue, num_verts*sizeof(bool)) == hipSuccess);
  assert(hipMallocManaged(&in_queue_next, num_verts*sizeof(bool)) == hipSuccess);
  hipMemcpy(queue_size, &init_queue_size, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(queue_next_size, &init_queue_size, sizeof(int), hipMemcpyHostToDevice);
  
  if (verbose) printf("Done: %lf (s)\n", omp_get_wtime() - elt);
  
  // move data to GPU
  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(g->out_adjlist, num_edges*sizeof(int), device, NULL);
  hipMemPrefetchAsync(g->out_offsets, (num_verts + 1)*sizeof(long), device, NULL);
  hipMemPrefetchAsync(t_parents, num_verts*sizeof(int), device, NULL);
  hipMemPrefetchAsync(t_levels, num_verts*sizeof(int), device, NULL);
  hipMemPrefetchAsync(f_parents, num_verts*sizeof(int), device, NULL);
  hipMemPrefetchAsync(f_levels, num_verts*sizeof(int), device, NULL);
  hipMemPrefetchAsync(labels, num_verts*sizeof(int), device, NULL);
  hipMemPrefetchAsync(queue, num_verts*sizeof(int), device, NULL);
  hipMemPrefetchAsync(queue_size, sizeof(int), device, NULL);
  hipMemPrefetchAsync(queue_next, num_verts*sizeof(int), device, NULL);
  hipMemPrefetchAsync(queue_next_size, sizeof(int), device, NULL);
  hipMemPrefetchAsync(in_queue, num_verts*sizeof(bool), device, NULL);
  hipMemPrefetchAsync(in_queue_next, num_verts*sizeof(bool), device, NULL);

  // Timer for no-gpu timing data
  elt = omp_get_wtime();

  // *********************************************************************************
  // run initial BFS to get spanning tree T
  // *********************************************************************************
  spanning_tree(g, root, t_parents, t_levels, queue, queue_size, queue_next, queue_next_size, in_queue, in_queue_next);

  // *********************************************************************************
  // Run ConnectedComponents on G - T
  // *********************************************************************************
  connected_components(g, t_parents, labels, queue, queue_size, queue_next, queue_next_size, in_queue, in_queue_next);

  // *********************************************************************************
  // run final BFS from sources in labels to get spanning forest F
  // *********************************************************************************
  spanning_forest(g, labels, t_parents, f_parents, f_levels, queue, queue_size, queue_next, queue_next_size, in_queue, in_queue_next);

  // *********************************************************************************
  // Update return values with T & F
  // *********************************************************************************
  add_new_edges(f_parents, t_parents, new_srcs, new_dsts, new_num_edges, num_verts);

  if (verbose) printf("\nFiltering Runtime W/O GPU Overhead: %lf (s)\n", omp_get_wtime() - elt);

  // *********************************************************************************
  // Free cuda memory
  // *********************************************************************************
  hipFree(g->out_adjlist);
  hipFree(g->out_offsets);
  hipFree(g);

  hipFree(labels);
  hipFree(root);
  hipFree(t_parents);
  hipFree(t_levels);
  hipFree(f_parents);
  hipFree(f_levels);

  hipFree(queue);
  hipFree(queue_size);
  hipFree(queue_next);
  hipFree(queue_next_size);
  hipFree(in_queue);
  hipFree(in_queue_next);

  return 1;
}
